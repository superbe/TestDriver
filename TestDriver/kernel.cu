// http://docs.nvidia.com/cuda/cuda-runtime-api/index.html#ixzz4kI0W7OiI

#include "hip/hip_runtime.h"
#include <iostream>

int main(int argc, char *argv[])
{
	setlocale(LC_ALL, "Russian");
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	printf("���������� ������������� ���������:                                                 %d\n", count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("--------------------------------------------------------------------------------", prop.name);
		printf("\n");
		printf("�������� �� ���������� %d\n", i);
		printf("\n");
		printf("������������:                                                                       %s\n", prop.name);
		printf("��������� (Compute capability):                                                     %d.%d\n", prop.major, prop.minor);
		printf("�������� ������� � ����������:                                                      %d\n", prop.clockRate);
		printf("���������� ����������� ������������ DMA:                                            %d\n", prop.asyncEngineCount);
		printf("����-��� ���������� ����:                                                           ");
		if (prop.kernelExecTimeoutEnabled) printf("�������\n"); else printf("��������\n");
		printf("���������� GPU �������������:                                                       ");
		if (prop.integrated) printf("��\n"); else printf("���\n");
		printf("����� ����������:                                                                   %d ", prop.computeMode);
		switch (prop.computeMode)
		{
		case 1:
			printf("(�������������� ����� ����������, ������ ���� ���� � ����� �������� ������ ������������ hipSetDevice())\n", prop.computeMode);
			break;
		case 2:
			printf("(����������� ����� ����������, �� ���� ���� �� ����� ������������ hipSetDevice())\n", prop.computeMode);
			break;
		case 3:
			printf("(���������� ����� ����������, ��������� ����� � ����� �������� ������ ������������ hipSetDevice())\n", prop.computeMode);
			break;
		default:
			printf("(����� ���������� �� ���������, ��������� ������� ����� ������������ hipSetDevice())\n", prop.computeMode);
		}
		printf("���������� ������������ ���������� ���������� ���� ������������ � ����� ���������:  ");
		if (prop.concurrentKernels) printf("��\n"); else printf("���\n");


		printf("\n");

		printf("�������� � ������ ���������� %d\n", i);
		printf("\n");
		printf("������ ���������� ������ � ������:                                                  %ld\n", prop.totalGlobalMem);
		printf("������ ����������� ������ � ������:                                                 %ld\n", prop.totalConstMem);
		printf("������������ ��� ����������� � ������:                                              %ld\n", prop.memPitch);
		printf("������������ �������:                                                               %ld\n", prop.textureAlignment);
		printf("�������� ���������� ������ CPU �� �������� ������������ Cuda-����������:            ");
		if (prop.canMapHostMemory) printf("��\n"); else printf("���\n");
		printf("\n");

		printf("�������� � ����������������� ���������� %d\n", i);
		printf("\n");
		printf("���������� �����������������:                                                       %d\n", prop.multiProcessorCount);
		printf("����������� ������ �� ���� ���������������:                                         %ld\n", prop.sharedMemPerBlock);
		printf("��������� �� ���� ���������������:                                                  %d\n", prop.regsPerBlock);
		printf("����� �� ����:                                                                      %d\n", prop.warpSize);
		printf("������������ ���������� ����� � �����:                                              %d\n", prop.maxThreadsPerBlock);
		printf("������������ ���������� ����� �� ����������:                                        (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("������������ ������� �����:                                                         (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");








		//printf("�������� ��������� ECC:  %d\n", prop.ECCEnabled);

			//printf(":  %d\n", prop.concurrentManagedAccess)
			//���������� ����� ���������� ������ � ����������� ������ ������������ � �����������

			//printf(":  %d\n", prop.globalL1CacheSupported)
			//���������� ������������ ����������� ������ � L1

			//printf(":  %d\n", prop.hostNativeAtomicSupported)
			//����� ����� ����������� � ������ ������������ ������ ��������� ��������

			//printf(":  %d\n", prop.isMultiGpuBoard)
			//���������� �� ���������� GPU �����

			//printf(":  %d\n", prop.l2CacheSize)
			//������ ���� L2 � ������

			//printf(":  %d\n", prop.localL1CacheSupported)
			//���������� ������������ ����������� ������� ������� � L1

			//printf(":  %d\n", prop.managedMemory)
			//������������ ���������� ������ �������� �� ����������� �� ���� �������

			//printf(":  %d\n", prop.maxSurface1D)
			//������������ ������ 1D �����������

			//��� hipDeviceProp_t::maxSurface1DLayered[2])
			//������������ ������� 1D �������� �����������

			//��� hipDeviceProp_t::maxSurface2D[2])
			//������������ ������� ����������� 2D

			//��� hipDeviceProp_t::maxSurface2DLayered[3])
			//������������ ������� �������� 2D �����������

			//��� hipDeviceProp_t::maxSurface3D[3])
			//������������ ������� ����������� 3D

			//printf(":  %d\n", prop.maxSurfaceCubemap)
			//������������ ������� ����������� Cubemap

			//��� hipDeviceProp_t::maxSurfaceCubemapLayered[2])
			//������������ ������� �������� Cubemap �����������

			//printf(":  %d\n", prop.maxTexture1D)
			//������������ ������ �������� 1D

			//��� hipDeviceProp_t::maxTexture1DLayered[2])
			//������������ ������� 1D �������� ��������

			//printf(":  %d\n", prop.maxTexture1DLinear)
			//������������ ������ 1D ������� ������� � �������� �������

			//printf(":  %d\n", prop.maxTexture1DMipmap)
			//������������ 1D ��� ������ ��������

			//��� hipDeviceProp_t::maxTexture2D[2])
			//������������ ������� 2D ��������

			//��� hipDeviceProp_t::maxTexture2DGather[2])
			//������������ ������� 2D ��������, ���� �������� ��������� �������� ������ ���� ���������

			//��� hipDeviceProp_t::maxTexture2DLayered[3])
			//������������ ������� 2D �������� ��������

			//��� hipDeviceProp_t::maxTexture2DLinear[3])
			//������������ �������(������, ������, ������) ��� 2D �������, ��������� � ������� �������

			//��� hipDeviceProp_t::maxTexture2DMipmap[2])
			//������������ ������� 2D ��� ��������

			//��� hipDeviceProp_t::maxTexture3D[3])
			//������������ ������� �������� 3D

			//��� hipDeviceProp_t::maxTexture3DAlt[3])
			//������������ ������� ���������� 3D ��������

			//printf(":  %d\n", prop.maxTextureCubemap)
			//������������ ������� �������� Cubemap

			//��� hipDeviceProp_t::maxTextureCubemapLayered[2])
			//������������ ������� Cubemap �������� ��������

			//printf(":  %d\n", prop.maxThreadsPerMultiProcessor)
			//������������ ������ ���������� �� �����������������

			//printf(":  %d\n", prop.memoryBusWidth)
			//������ ���� ���������� ������ � �����

			//printf(":  %d\n", prop.multiGpuBoardGroupID)
			//���������� ������������� ��� ������ ��������� �� ����� � ��� �� ������ - GPU �����

			//printf(":  %d\n", prop.pageableMemoryAccess)
			//���������� ������������ ���������� ������ � ����������� ������ ��� ������ hipHostRegister �� ���

			//printf(":  %d\n", prop.pciBusID)
			//���� PCI ������������� ����������

			//printf(":  %d\n", prop.pciDeviceID)
			//PCI - ���������� ������������� ����������

			//printf(":  %d\n", prop.pciDomainID)
			//PCI - ������������� ������ ����������

			//printf(":  %d\n", prop.regsPerMultiprocessor)
			//32 - ������ ���������, ��������� � �����������������

			//size_t hipDeviceProp_t::sharedMemPerMultiprocessor)
			//����� ������, ��������� �� ����������������� � ������

			//printf(":  %d\n", prop.singleToDoublePrecisionPerfRatio)
			//����������� ������������������ ����� ��������(� ��������� � ��������� ������� � �������) �� ������� �������� ����������

			//printf(":  %d\n", prop.streamPrioritiesSupported)
			//���������� ������������ ���������� �������

			//size_t hipDeviceProp_t::surfaceAlignment)
			//������������ ���������� � �����������

			//printf(":  %d\n", prop.tccDriver)
			//1, ���� ���������� �������� ����������� �����, ��������� ������� TCC, 0 � ��������� ������

			//size_t hipDeviceProp_t::texturePitchAlignment)
			//��� ���������� ������������ ��� ������ ������� ������� � ������� �������

			//printf(":  %d\n", prop.unifiedAddressing)
			//���������� ��������� ������ �������� ������������ � ��������


		//printf(":  %d\n", prop.concurrentManagedAccess);
		//printf(":  %d\n", prop.ECCEnabled);
		//printf(":  %d\n", prop.globalL1CacheSupported);
		//printf(":  %d\n", prop.hostNativeAtomicSupported);
		//printf(":  %d\n", prop.isMultiGpuBoard);
		//printf(":  %d\n", prop.l2CacheSize);
		//printf(":  %d\n", prop.localL1CacheSupported);
		//printf(":  %d\n", prop.managedMemory);

		//printf(":  %d\n", prop.maxSurface1D);
		//printf(":  %d\n", prop.maxSurface1DLayered);
		//printf(":  %d\n", prop.maxSurface2D);
		//printf(":  %d\n", prop.maxSurface2DLayered);
		//printf(":  %d\n", prop.maxSurface3D);
		//printf(":  %d\n", prop.maxSurfaceCubemap);
		//printf(":  %d\n", prop.maxSurfaceCubemapLayered);

		//printf(":  %d\n", prop.maxTexture1D);
		//printf(":  %d\n", prop.maxTexture1DLayered);
		//printf(":  %d\n", prop.maxTexture1DLinear);
		//printf(":  %d\n", prop.maxTexture1DMipmap);
		//printf(":  %d\n", prop.maxTexture2D);
		//printf(":  %d\n", prop.maxTexture2DGather);
		//printf(":  %d\n", prop.maxTexture2DLayered);
		//printf(":  %d\n", prop.maxTexture2DLinear);
		//printf(":  %d\n", prop.maxTexture2DMipmap);
		//printf(":  %d\n", prop.maxTexture3D);
		//printf(":  %d\n", prop.maxTexture3DAlt);
		//printf(":  %d\n", prop.maxTextureCubemap);
		//printf(":  %d\n", prop.maxTextureCubemapLayered);

		//printf(":  %d\n", prop.maxThreadsPerMultiProcessor);
		//printf(":  %d\n", prop.memoryBusWidth);
		//printf(":  %d\n", prop.multiGpuBoardGroupID);
		//printf(":  %d\n", prop.pageableMemoryAccess);
		//printf(":  %d\n", prop.pciBusID);
		//printf(":  %d\n", prop.pciDeviceID);
		//printf(":  %d\n", prop.pciDomainID);
		//printf(":  %d\n", prop.regsPerMultiprocessor);
		//printf(":  %d\n", prop.sharedMemPerMultiprocessor);
		//printf(":  %d\n", prop.singleToDoublePrecisionPerfRatio);
		//printf(":  %d\n", prop.streamPrioritiesSupported);
		//printf(":  %d\n", prop.surfaceAlignment);
		//printf(":  %d\n", prop.tccDriver);
		//printf(":  %d\n", prop.texturePitchAlignment);
		//printf(":  %d\n", prop.unifiedAddressing);
	}

	system("pause");

	return 0;
}